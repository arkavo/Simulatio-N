#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include "Vector.h"
#include "Sequence.h"

#define size 100000;                                    //space size
#define t 0.0001;                                       //timeStep

using namespace std;

float Field[size][size][size]={0};


__global__ void Movement(planet* pl,vec force)
{
  pl->pos.x1 += (*pl).vel.x1 * t;
  pl->pos.x2 += (*pl).vel.x2 * t;
  pl->pos.x3 += (*pl).vel.x3 * t;

  pl->vel.x1 += ((force.x1)/((*pl).mass)) * t;
  pl->vel.x2 += ((force.x2)/((*pl).mass)) * t;
  pl->vel.x3 += ((force.x3)/((*pl).mass)) * t;
}


int main(void)
{
  int n;                                               //planet counter
  int i=0;                                             //counter variable
  int j;
  int* t_mass;
  vec* t_pos;


  cout<<"Input number of bodies:\n";
  cin>>n;
  Body* List[n];
  for(i=0;i<n;i++)
  {
      List[i]=hipMalloc((void**)&Body,sizeof(Body));
      List[i]->vel = {0};
      cout<<"mass and postion of"<<i+1<<"th Body?\n";
      cin>>List[i]->mass>>List[i]->x>>List[i]->y>>List[i]->z;
      hipMemcpy(t_mass,List[i].mass,sizeof(float),hipMemcpyHostToDevice);
      for(j=0;j<3;j++)
        hipMemcpy(t_pos[j],List[i].pos[j],sizeof(float),hipMemcpyHostToDevice);
  }
}
